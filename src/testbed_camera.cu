#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>

#include <neural-graphics-primitives/testbed.h>

NGP_NAMESPACE_BEGIN

void Testbed::init_camera() {
    cfg.enable_stream(RS2_STREAM_COLOR, color_width, color_height, RS2_FORMAT_BGR8, 30);
    pipe.start(cfg);
    intrinsics = pipe.get_active_profile()
                    .get_stream(RS2_STREAM_COLOR)
                    .as<rs2::video_stream_profile>()
                    .get_intrinsics(); 
    cameraMatrix = cv::Mat::eye(3, 3, CV_64F);
    cameraMatrix.at<double>(0, 0) = intrinsics.fx;
    cameraMatrix.at<double>(1, 1) = intrinsics.fy;
    cameraMatrix.at<double>(0, 2) = intrinsics.ppx;
    cameraMatrix.at<double>(1, 2) = intrinsics.ppy;

}

void Testbed::get_color_image(){
    frames = pipe.wait_for_frames();
    color = frames.get_color_frame();
    colorImage = cv::Mat(cv::Size(color_width, color_height), CV_8UC3, (void*)color.get_data(), cv::Mat::AUTO_STEP);
}

void Testbed::set_aruco_board(){
    board = cv::aruco::GridBoard::create(5, 7, markerLength, gap, arucoDict);
}

void Testbed::get_aruco_pose(){
    outputImage = colorImage.clone();
    cv::aruco::detectMarkers(outputImage, arucoDict, markerCorners, markerIds);
    if(markerIds.size() > 0){
        cv::aruco::drawDetectedMarkers(outputImage, markerCorners, markerIds);
        cv::aruco::estimatePoseBoard(markerCorners, markerIds, board, cameraMatrix, distCoeffs, rvec, tvec);
        cv::Rodrigues(rvec, R);
        const double* dataPtr = R.ptr<double>(0);
        for(int i = 0; i < 3; ++i){
            for(int j = 0; j < 3; ++j){
                cameraPose[j][i] = dataPtr[i * 3 + j];
            }
        }
        dataPtr = tvec.ptr<double>(0);
        for(int i = 0; i < 3; ++i){
            cameraPose[3][i] = dataPtr[i];
        }
        isaruco = true;
    } else {
        isaruco = false;
    }
}

void Testbed::color_to_texture(){
    glGenTextures(1, &aruco_texture);
    glBindTexture(GL_TEXTURE_2D, aruco_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glPixelStorei(GL_UNPACK_ROW_LENGTH, 0);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, outputImage.cols, outputImage.rows, 0, GL_BGR, GL_UNSIGNED_BYTE, outputImage.ptr());
}

void Testbed::visualize_camera_pose(ImDrawList* list, const mat4& world2proj){
    ivec2 res{color_width, color_height};
    float aspect = float(res.x) / float(res.y);
    visualize_nerf_camera(list, world2proj, cameraPose, aspect, isaruco ? 0x48ff00ff : 0xfc050dff);
}


void Testbed::gen_candidate_views(){
    // define theta phi to xyz function using lambda
    auto theta_phi_to_xyz = [](float theta, float phi, float radius) -> glm::vec3 {
    float x = radius * sin(theta) * cos(phi);
    float y = radius * sin(theta) * sin(phi);
    float z = radius * cos(theta);
    return glm::vec3(x, y, z);
};

    vector<float> thetas(n_steps);
    vector<float> radiuses(n_steps);
    vector<int> n_view_in_steps(n_steps);

    // calculate thetas and radiuses
    for (int i = 0; i < num_steps; ++i) {
        thetas[i] = (M_PI / 2) * static_cast<float>(i) / num_steps;
        radiuses[i] = sin(thetas[i]);
    }

    float radius_sum = std::accumulate(radiuses.begin(), radiuses.end(), 0.0f);
    for (int i = 0; i < n_steps; ++i) {
        n_view_in_steps[i] = std::round(n_points * radiuses[i] / radius_sum);
    }

    // generate points and rotation matrices
    for (int i = 0; i < n_steps; ++i) {
        for (int j = 0; j < n_view[i]; ++j) {
            float phi = 2 * M_PI * static_cast<float>(j) / n_view[i];
            vec3 tvec = theta_phi_to_xyz(thetas[i], phi, radius) + origin;

            // calculate orientation
            vec3 u_z = -1.0f * normalize(theta_phi_to_xyz(thetas[i], phi, 1));
            vec3 u_y = normalize(theta_phi_to_xyz(thetas[i] + M_PI / 2, phi, 1));
            vec3 u_x = normalize(cross(u_y, u_z));

            // rotation matrix
            mat3 rmat = inverse(mat3(u_x, u_y, u_z));

            // Convert rotation matrix to quaternion
            quat tmp_quat = quat_cast(rmat);

            // Get angle and axis quaternion
            float angle = angle(tmp_quat);
            vec3 axis = axis(tmp_quat);

            // Rotation vector is angle * axis
            vec3 rvec = angle * axis;

            m_candidate_views.push_back({tvec, rvec});
        }
    }
}

NGP_NAMESPACE_END