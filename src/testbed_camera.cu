#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>

#include <neural-graphics-primitives/testbed.h>


NGP_NAMESPACE_BEGIN

bool Testbed::init_camera() {
    if (cam_type == CameraType::NOCAM) {
        return false;
    }
    else if (cam_type == CameraType::USB) {
        // something about usb camera config

        return true;
    }
    else if (cam_type == CameraType::REALSENSE) {
        // something about realsense config
        cfg.enable_stream(RS2_STREAM_COLOR, color_width, color_height, RS2_FORMAT_BGR8, 30);
        pipe.start(cfg);
        intrinsics = pipe.get_active_profile()
                        .get_stream(RS2_STREAM_COLOR)
                        .as<rs2::video_stream_profile>()
                        .get_intrinsics(); 
        cameraMatrix = cv::Mat::eye(3, 3, CV_64F);
        cameraMatrix.at<double>(0, 0) = intrinsics.fx;
        cameraMatrix.at<double>(1, 1) = intrinsics.fy;
        cameraMatrix.at<double>(0, 2) = intrinsics.ppx;
        cameraMatrix.at<double>(1, 2) = intrinsics.ppy;

        return false;
    }
    else {
        return false;
    }
}

void Testbed::get_color_image(){
    frames = pipe.wait_for_frames();
    color = frames.get_color_frame();
    colorImage = cv::Mat(cv::Size(color_width, color_height), CV_8UC3, (void*)color.get_data(), cv::Mat::AUTO_STEP);
}

void Testbed::set_aruco_board(){
    board = cv::aruco::GridBoard::create(5, 7, markerLength, gap, arucoDict);
}

void Testbed::get_aruco_pose(){
    outputImage = colorImage.clone();
    cv::aruco::detectMarkers(outputImage, arucoDict, markerCorners, markerIds);
    if(markerIds.size() > 0){
        cv::aruco::drawDetectedMarkers(outputImage, markerCorners, markerIds);
        cv::aruco::estimatePoseBoard(markerCorners, markerIds, board, cameraMatrix, distCoeffs, rvec, tvec);
        cv::Rodrigues(rvec, R);
        const double* dataPtr = R.ptr<double>(0);
        for(int i = 0; i < 3; ++i){
            for(int j = 0; j < 3; ++j){
                cameraPose[j][i] = dataPtr[i * 3 + j];
            }
        }
        dataPtr = tvec.ptr<double>(0);
        for(int i = 0; i < 3; ++i){
            cameraPose[3][i] = dataPtr[i];
        }
        isaruco = true;
    } else {
        isaruco = false;
    }
}

void Testbed::color_to_texture(){
    glGenTextures(1, &aruco_texture);
    glBindTexture(GL_TEXTURE_2D, aruco_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glPixelStorei(GL_UNPACK_ROW_LENGTH, 0);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, outputImage.cols, outputImage.rows, 0, GL_BGR, GL_UNSIGNED_BYTE, outputImage.ptr());
}

void Testbed::visualize_camera_pose(ImDrawList* list, const mat4& world2proj){
    ivec2 res{color_width, color_height};
    float aspect = float(res.x) / float(res.y);
    visualize_nerf_camera(list, world2proj, cameraPose, aspect, isaruco ? 0x48ff00ff : 0xfc050dff);
}


void Testbed::gen_candidate_views(){
    // define theta phi to xyz function using lambda
    auto theta_phi_to_xyz = [](float theta, float phi, float radius) -> glm::vec3 {
        float x = radius * sin(theta) * cos(phi);
        float y = radius * sin(theta) * sin(phi);
        float z = radius * cos(theta);
        return glm::vec3(x, y, z);
    };

    // we can define thetas, radiuses, n_view_in_steps which are depenedent on n_steps
    std::vector<float> thetas(n_steps);
    std::vector<float> radiuses(n_steps);
    std::vector<int> n_view_in_steps(n_steps); // number of views in each step

    // calculate thetas and radiuses
    for (int i = 0; i < n_steps; i++) {
        thetas[i] = (M_PI / 2) * static_cast<float>(i) / n_steps;
        radiuses[i] = sin(thetas[i]);
    }

    float radius_sum = std::accumulate(radiuses.begin(), radiuses.end(), 0.0f);
    for (int i = 0; i < n_steps; i++) {
        n_view_in_steps[i] = std::round(n_points * radiuses[i] / radius_sum);
    }

    // generate points and rotation matrices
    for (int i = 0; i < n_steps; i++) {
        for (int j = 0; j < n_view_in_steps[i]; j++) {
            float phi = 2 * M_PI * static_cast<float>(j) / n_view_in_steps[i];
            vec3 tvec = theta_phi_to_xyz(thetas[i], phi, radius) + origin;

            // calculate orientation
            vec3 u_z = -1.0f * normalize(theta_phi_to_xyz(thetas[i], phi, 1));
            vec3 u_y = normalize(theta_phi_to_xyz(thetas[i] + M_PI / 2, phi, 1));
            vec3 u_x = normalize(cross(u_y, u_z));

            // rotation matrix
            mat3 rmat = inverse(mat3(u_x, u_y, u_z));

            // Convert rotation matrix to quaternion
            quat tmp_quat = quat_cast(rmat);

            mat4 tmp_mat = glm::translate(glm::toMat4(tmp_quat), tvec);
            
            // CandidateView tmp_cv{tvec, tmp_quat, tmp_mat};
            m_candidate_views.push_back({tvec, tmp_quat, tmp_mat});
        }
    }
}

// load cam, board, candidate view cfgs
void Testbed::load_mint_config(const fs::path& path) {    
    std::ifstream f(native_string(path));
    nlohmann::json data = nlohmann::json::parse(f);

    nlohmann::json cam_cfg = data["camera"];
    nlohmann::json board_cfg = data["board"];
    nlohmann::json can_cfg = data["candidate_view"];

    tlog::info() << "Start candidate handler initialize";
    candidate_handler.init(can_cfg);
    tlog::info() << "End candidate handler initialize";

    tlog::info() << "type value is " << cam_cfg["type"];
    CameraType cam_type = (CameraType) cam_cfg["type"];
    switch (cam_type) {
        case CameraType::USB:
            cam_stream = std::make_unique<Testbed::USBCamera>();
            break;
        case CameraType::REALSENSE:
            cam_stream = std::make_unique<Testbed::RealSense>();
            break;
    }

    cam_stream->init(cam_cfg);
}


void Testbed::SphereCandidates::init(nlohmann::json& j){
    radius = j["radius"];
    n_total_candidates = j["n_total_candidates"];
    n_floor = j["n_floor"];
    n_areas = j["n_areas"];

    tlog::info() << "radius " << radius;

}

void Testbed::SphereCandidates::gen_candidate_views() {
    // define theta phi to xyz function using lambda
    auto theta_phi_to_xyz = [](float theta, float phi, float radius) -> glm::vec3 {
        float x = radius * sin(theta) * cos(phi);
        float y = radius * sin(theta) * sin(phi);
        float z = radius * cos(theta);
        return glm::vec3(x, y, z);
    };

    // we can define thetas, radiuses, n_view_in_steps which are depenedent on n_steps
    std::vector<float> thetas(n_floor);
    std::vector<float> radiuses(n_floor);
    std::vector<int> n_view_in_steps(n_floor); // number of views in each step

    // calculate thetas and radiuses
    for (int i = 0; i < n_floor; i++) {
        thetas[i] = (M_PI / 2) * static_cast<float>(i) / n_floor;
        radiuses[i] = sin(thetas[i]);
    }

    float radius_sum = std::accumulate(radiuses.begin(), radiuses.end(), 0.0f);
    for (int i = 0; i < n_floor; i++) {
        n_view_in_steps[i] = std::round(n_total_candidates * radiuses[i] / radius_sum);
    }

    // generate points and rotation matrices
    for (int i = 0; i < n_floor; i++) {
        for (int j = 0; j < n_view_in_steps[i]; j++) {
            float phi = 2 * M_PI * static_cast<float>(j) / n_view_in_steps[i];
            vec3 tvec = theta_phi_to_xyz(thetas[i], phi, radius) + origin;

            // calculate orientation
            vec3 u_z = -1.0f * normalize(theta_phi_to_xyz(thetas[i], phi, 1));
            vec3 u_y = normalize(theta_phi_to_xyz(thetas[i] + M_PI / 2, phi, 1));
            vec3 u_x = normalize(cross(u_y, u_z));

            // rotation matrix
            mat3 rmat = inverse(mat3(u_x, u_y, u_z));

            // Convert rotation matrix to quaternion
            quat tmp_quat = quat_cast(rmat);

            // Make Transform matrix
            mat4 tmp_mat = glm::translate(glm::toMat4(tmp_quat), tvec);
            
            // CandidateView tmp_cv{tvec, tmp_quat, tmp_mat};
            v_candidate_views.push_back({tvec, tmp_quat, tmp_mat});
        }
    }
}


void Testbed::CameraStream::init(nlohmann::json& j) {
    // set cam type
    cam_type = (CameraType)j["type"];

    // set cam model
    model = j["model"];

    // set fps
    fps = j["fps"];

    // set resolution
    resolution.width = j["resolution"][0];
    resolution.height = j["resolution"][1];

    // set intrinsic
    intrinsic.at<double>(0) = j["intrinsic"][0];
    intrinsic.at<double>(1) = j["intrinsic"][1];
    intrinsic.at<double>(2) = j["intrinsic"][2];
    intrinsic.at<double>(3) = j["intrinsic"][3];

    // set cameraMatrix
    camera_matrix.at<double>(0,0) = j["intrinsic"][0];
    camera_matrix.at<double>(1,1) = j["intrinsic"][1];
    camera_matrix.at<double>(0,2) = j["intrinsic"][2];
    camera_matrix.at<double>(1,2) = j["intrinsic"][3];

    // set distort coefficient
    dist_coeff.at<double>(0) = j["dist_coeff"][0];
    dist_coeff.at<double>(1) = j["dist_coeff"][1];
    dist_coeff.at<double>(2) = j["dist_coeff"][2];
    dist_coeff.at<double>(3) = j["dist_coeff"][3];
    dist_coeff.at<double>(4) = j["dist_coeff"][4];    
}

void Testbed::USBCamera::init(nlohmann::json& j) {
    // init using camera stream
    CameraStream::init(j);

    // open usb cap
    device = j["device"];
    cap.open(device.c_str());
}

void Testbed::RealSense::init(nlohmann::json& j) {
    // init using camera stream
    CameraStream::init(j);

    cfg.enable_stream(RS2_STREAM_COLOR, resolution.width, resolution.height, RS2_FORMAT_BGR8, fps);
    pipe.start(cfg);
}

bool Testbed::USBCamera::get_color_image(cv::Mat& color_image){
    if (!cap.isOpened())
    {
        printf("Can't open the video");
        return false;
    }

    cap >> color_image;
    return true;
}

bool Testbed::RealSense::get_color_image(cv::Mat& color_image) {
    frames = pipe.wait_for_frames();
    color = frames.get_color_frame();
    
    color_image = cv::Mat(resolution, CV_8UC3, (void*)color.get_data(), cv::Mat::AUTO_STEP);
    return true; // there must be something bad case. make it clear and add false return case
}


NGP_NAMESPACE_END